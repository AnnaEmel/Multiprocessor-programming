
#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <chrono>

typedef unsigned int uint;

void GenInput(int size, int num_steps);

void Process_GPU( bool* field_in, bool* field_out, uint field_size, uint num_steps );
__global__ void ProcessCell_GPU( bool* field_in, bool* field_out, int field_size );

void Process_CPU( bool* field_in, bool* field_out, uint field_size, uint num_steps );
void ProcessCell_CPU( bool* field_in, bool* field_out, uint i, int field_size );

void WriteResults( bool* out, uint field_size, const std::string& name );

int main(int argc, char* argv[])
{
	GenInput(100, 500);

	std::ifstream f( "input.txt" );
	if( !f )
	{
		std::cout << "Cannot open 'input.txt'" << std::endl;
		return -1;
	}

	uint field_size = 0;
	f >> field_size;

	uint num_steps = 0;
	f >> num_steps;

	uint cells_count = field_size*field_size;

	bool* field_in = new bool[cells_count];

	for( uint i = 0; i < cells_count; ++i )
		f >> field_in[i];
	
	bool* field_out_gpu = new bool[cells_count];
	bool* field_out_cpu = new bool[cells_count];

	std::chrono::time_point<std::chrono::steady_clock> t;
	std::chrono::microseconds delta;

	std::cout << "Starting GPU simulation..." << std::endl;
	t = std::chrono::steady_clock::now();
	Process_GPU( field_in, field_out_gpu, field_size, num_steps );
	delta = std::chrono::duration_cast<std::chrono::microseconds>( std::chrono::steady_clock::now() - t );
	std::cout << "GPU time: " << delta.count() << " microseconds" << std::endl;

	std::cout << "----------------------------------------" << std::endl;

	std::cout << "Starting CPU single thread simulation..." << std::endl;
	t = std::chrono::steady_clock::now();
	Process_CPU( field_in, field_out_cpu, field_size, num_steps );
	delta = std::chrono::duration_cast<std::chrono::microseconds>( std::chrono::steady_clock::now() - t );
	std::cout << "CPU time: " << delta.count() << " microseconds" << std::endl;

	for( uint i = 0; i < cells_count; ++i )
		if( field_out_cpu[i] != field_out_gpu[i] )
		{
			std::cout << "Validation fail" << std::endl;
			return -1;
		}

	WriteResults( field_out_cpu, field_size, "result.txt" );

	delete[] field_in;
	delete[] field_out_gpu;
	delete[] field_out_cpu;

	return 0;
}

void GenInput(int size, int num_steps)
{
	std::ofstream f( "input.txt" );

	f << size << ' ' << num_steps << '\n';
	for( int i = 0; i < size; ++i )
	{
		for( int j = 0; j < size; ++j )
		{
			f << rand() % 2 << ' ';
		}
		f << '\n';
	}
}

void Process_GPU( bool* field_in, bool* field_out, uint field_size, uint num_steps )
{
	uint cells_count = field_size*field_size;
	size_t array_size = sizeof( bool ) * cells_count;

	bool* fields[2];

	hipMalloc( (void**)( &fields[0] ), array_size );
	hipMalloc( (void**)( &fields[1] ), array_size );

	hipMemcpy( fields[0], field_in, array_size, hipMemcpyHostToDevice );

	const int threads_per_block = 100;

	int in, out;

	for( uint i = 0; i < num_steps; ++i )
	{
		in = i % 2;
		out = ( i + 1 ) % 2;
		ProcessCell_GPU <<< cells_count / threads_per_block, threads_per_block >>> ( fields[in], fields[out], field_size );
		hipMemcpy( fields[in], fields[out], array_size, hipMemcpyDeviceToDevice );
	}

	hipMemcpy( field_out, fields[in], array_size, hipMemcpyDeviceToHost );

	hipFree( fields[0] );
	hipFree( fields[1] );
}

__global__ void ProcessCell_GPU( bool* field_in, bool* field_out, int field_size )
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;

	int x = i % field_size;
	int y = i / field_size;

	int neighbours_count = 0;

	for( int cx = x - 1; cx <= x + 1; ++cx )
		for( int cy = y - 1; cy <= y + 1; ++cy )
		{
			if( cx == x && cy == y )
				continue;

			int nx = cx;
			int ny = cy;

			if( nx < 0 )
				nx = field_size - 1;

			if( nx >= field_size )
				nx = 0;

			if( ny < 0 )
				ny = field_size - 1;

			if( ny >= field_size )
				ny = 0;

			neighbours_count += (int)field_in[nx + ny * field_size];
		}

	if( field_in[i] )
	{
		if( neighbours_count == 2 || neighbours_count == 3 )
			field_out[i] = true;
		else
			field_out[i] = false;
	}
	else
	{
		if( neighbours_count == 3 )
			field_out[i] = true;
		else
			field_out[i] = false;
	}
}

void Process_CPU( bool* field_in, bool* field_out, uint field_size, uint num_steps )
{
	uint cells_count = field_size*field_size;
	size_t array_size = sizeof( bool ) * cells_count;

	bool* fields[2];
	fields[0] = new bool[cells_count];
	fields[1] = new bool[cells_count];

	memcpy( fields[0], field_in, array_size );

	int in, out;

	for( uint i = 0; i < num_steps; ++i )
	{
		in = i % 2;
		out = ( i + 1 ) % 2;
		for( uint j = 0; j < cells_count; ++j )
			ProcessCell_CPU( fields[in], fields[out], j, field_size );

		memcpy( fields[in], fields[out], array_size );
	}

	memcpy( field_out, fields[out], array_size );

	delete[] fields[0];
	delete[] fields[1];
}

void ProcessCell_CPU( bool* field_in, bool* field_out, uint i, int field_size )
{
	int x = i % field_size;
	int y = i / field_size;

	int neighbours_count = 0;

	for( int cx = x - 1; cx <= x + 1; ++cx )
		for( int cy = y - 1; cy <= y + 1; ++cy )
		{
			if( cx == x && cy == y )
				continue;

			int nx = cx;
			int ny = cy;

			if( nx < 0 )
				nx = field_size - 1;

			if( nx >= field_size )
				nx = 0;

			if( ny < 0 )
				ny = field_size - 1;

			if( ny >= field_size )
				ny = 0;

			neighbours_count += (int)field_in[nx + ny * field_size];
		}

	if( field_in[i] )
	{
		if( neighbours_count == 2 || neighbours_count == 3 )
			field_out[i] = true;
		else
			field_out[i] = false;
	}
	else
	{
		if( neighbours_count == 3 )
			field_out[i] = true;
		else
			field_out[i] = false;
	}
}

void WriteResults( bool* out, uint field_size, const std::string& name )
{
	std::ofstream of( name );

	for( uint i = 0; i < field_size; ++i )
	{
		for( uint j = 0; j < field_size; ++j )
			of << out[i * field_size + j] << ' ';

		of << '\n';
	}
}